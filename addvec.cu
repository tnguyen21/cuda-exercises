
#include <hip/hip_runtime.h>
#include <stdio.h>

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
  for (int i=0; i<n; ++i) {
      C_h[i] = A_h[i] + B_h[i];
    }
}

__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i < n) { //init
    C[i] = A[i] + B[i];
  }
}

void cuVecAdd(float* A, float* B, float* C, int n) {
  int size = n*sizeof(float);
  float *A_d, *B_d, *C_d;

  hipMalloc((void **) &A_d, size);
  hipMalloc((void **) &B_d, size);
  hipMalloc((void **) &C_d, size);

  hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

  // call kernel
  vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);
  
  hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main() {
  int n = 256;
  float A[n], B[n], C[n];

  for (int i=0; i<n; ++i) {
    A[i] = 1.1;
    B[i] = 2.3;
    C[i] = 0.0;
  }
  
  cuVecAdd(A, B, C, n);

  for (int i=0; i<n; ++i) {
    printf("C[%d] = %f\n", i, C[i]);
  }
 
  return 0;
} 
